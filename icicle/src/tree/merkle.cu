#include "hip/hip_runtime.h"
#include "hash/hash.cuh"
#include "merkle-tree/merkle.cuh"

namespace merkle_tree {
  /// Flattens the tree digests and sum them up to get
  /// the memory needed to contain all the digests
  template <typename S>
  size_t get_digests_len(uint32_t height, uint32_t arity)
  {
    size_t digests_len = 0;
    size_t row_length = 1;
    for (int i = 0; i < height; i++) {
      digests_len += row_length;
      row_length *= arity;
    }

    return digests_len;
  }

  /// Constructs merkle subtree without parallelization
  /// The digests are aligned sequentially per row
  /// Example:
  ///
  /// Big tree:
  ///
  ///        1      <- Root
  ///       / \     <- Arity = 2
  ///      2   3    <- Digests
  ///     / \ / \   <- Height = 2 (as the number of edges)
  ///    4  5 6  7  <- height^arity leaves
  ///    |  | |  |  <- Sponge hash 1 to 1
  ///    a  b c  d  <- Input vector 1x4
  ///
  /// Subtree 1    Subtree 2
  ///    2            3
  ///   / \          / \
  ///  4   5        6   7
  ///
  /// Digests array for subtree 1:
  /// [4 5 . . 2 . .]
  /// |   |    |
  /// -----    V
  ///   |    Segment (offset = 4, subtree_idx = 0)
  ///   v
  /// Segment (offset = 0, subtree_idx = 0)
  ///
  /// Digests array for subtree 2:
  /// [. . 6 7 . 3 .]
  ///     |   |
  ///     -----
  ///       |
  ///       v
  ///    Segment (offset = 0, subtree_idx = 1)
  ///
  /// Total digests array:
  /// [4 5 6 7 2 3 .]
  template <typename L, typename D>
  hipError_t build_merkle_subtree(
    L* leaves,
    D* states,
    D* digests,
    size_t subtree_idx,
    size_t subtree_height,
    L* big_tree_digests,
    size_t start_segment_size,
    size_t start_segment_offset,
    int keep_rows,
    const SpongeHasher<L, D>& sponge,
    const CompressionHasher<D>& compression,
    const SpongeConfig& sponge_config,
    hipStream_t& stream)
  {
    unsigned int arity = compression.preimage_max_length;

    size_t leaves_size = pow(arity, subtree_height);

    sponge.absorb_many(leaves, states, leaves_size, sponge_config);
    sponge.squeeze_many(states, digests, leaves_size, sponge_config);

    uint32_t number_of_states = leaves_size;
    size_t segment_size = start_segment_size;
    size_t segment_offset = start_segment_offset;

    device_context::DeviceContext subtree_ctx{
      stream,
      sponge_config.ctx.device_id,
      sponge_config.ctx.mempool,
    };

    while (number_of_states > 0) {
      hipError_t compression_error =
        compression.compress_many(digests, digests, number_of_states, sponge_config.offset, states, subtree_ctx, true);
      CHK_IF_RETURN(compression_error);

      if (!keep_rows || subtree_height <= keep_rows) {
        S* digests_with_offset = big_tree_digests + segment_offset + subtree_idx * number_of_states;
        CHK_IF_RETURN(
          hipMemcpyAsync(digests_with_offset, digests, number_of_states * sizeof(D), hipMemcpyDeviceToHost, stream));
        segment_offset += segment_size;
      }

      segment_size /= arity;
      subtree_height--;
      number_of_states /= arity;
      // config.aligned = true;
    }

    return CHK_LAST();
  }

  template <typename L, typename D>
  hipError_t build_merkle_tree(
    const L* leaves,
    D* digests,
    uint32_t height,
    const SpongeHasher<L, D>& sponge,
    const CompressionHasher<D>& compression,
    const SpongeConfig& sponge_config,
    const TreeBuilderConfig& tree_config)
  {
    CHK_INIT_IF_RETURN();
    hipStream_t& stream = config.ctx.stream;

    if (sponge_config.input_block_len >= sponge.preimage_max_length)
      THROW_ICICLE_ERR(
        IcicleError_t::InvalidArgument,
        "Sponge construction at the bottom of the tree doesn't support inputs bigger than the size of the state");
    if (sponge_config.output_len != 1)
      THROW_ICICLE_ERR(
        IcicleError_t::InvalidArgument,
        "Sponge construction at the bottom of the tree should have an output len of 1 element");
    if (compression.preimage_max_length != tree_config.arity)
      THROW_ICICLE_ERR(IcicleError_t::InvalidArgument, "Hash max preimage length does not match merkle tree arity");

    uint32_t number_of_leaves = pow(tree_config.arity, height);
    uint32_t total_number_of_leaves = number_of_leaves * sponge_config.input_block_len;

    // This will determine how much splitting do we need to do
    // `number_of_streams` subtrees should fit in the device
    // This means each subtree should fit in `STREAM_CHUNK_SIZE` memory
    uint32_t number_of_subtrees = 1;
    uint32_t subtree_height = height;
    uint32_t subtree_leaves_size = number_of_leaves;
    uint32_t subtree_states_size = subtree_leaves_size * sponge.width;
    uint32_t subtree_digests_size = subtree_leaves_size;
    size_t subtree_memory_required = sizeof(D) * (subtree_states_size + subtree_digests_size);
    while (subtree_memory_required > STREAM_CHUNK_SIZE) {
      number_of_subtrees *= tree_config.arity;
      subtree_height--;
      subtree_leaves_size /= tree_config.arity;
      subtree_states_size /= tree_config.arity;
      subtree_digests_size /= tree_config.arity;
      subtree_memory_required = sizeof(S) * (subtree_states_size + subtree_digests_size);
    }
    int cap_height = height - subtree_height;
    size_t caps_len = pow(tree_config.arity, cap_height);

    size_t available_memory, _total_memory;
    CHK_IF_RETURN(hipMemGetInfo(&available_memory, &_total_memory));
    available_memory -= GIGA / 8; // Leave 128 MB just in case

    // We can effectively parallelize memory copy with streams
    // as long as they don't operate on more than `STREAM_CHUNK_SIZE` bytes
    const size_t number_of_streams = std::min((uint32_t)(available_memory / STREAM_CHUNK_SIZE), number_of_subtrees);
    hipStream_t* streams = static_cast<hipStream_t*>(malloc(sizeof(hipStream_t) * number_of_streams));
    for (size_t i = 0; i < number_of_streams; i++) {
      CHK_IF_RETURN(hipStreamCreate(&streams[i]));
    }

    // Allocate memory for the leaves and digests
    // These are shared by streams in a pool
    D *states_ptr, *digests_ptr;
    CHK_IF_RETURN(hipMallocAsync(&states_ptr, subtree_states_size * number_of_streams * sizeof(D), stream))
    CHK_IF_RETURN(hipMallocAsync(&digests_ptr, subtree_digests_size * number_of_streams * sizeof(D), stream))
    // Wait for these allocations to finish
    CHK_IF_RETURN(hipStreamSynchronize(stream));

    bool caps_mode = tree_config.keep_rows && tree_config.keep_rows < cap_height;
    D* caps;
    if (caps_mode) { caps = static_cast<D*>(malloc(caps_len * sizeof(D))); }

#if !defined(__CUDA_ARCH__) && defined(MERKLE_DEBUG)
    std::cout << "Available memory = " << available_memory / 1024 / 1024 << " MB" << std::endl;
    std::cout << "Number of streams = " << number_of_streams << std::endl;
    std::cout << "Number of subtrees = " << number_of_subtrees << std::endl;
    std::cout << "Height of a subtree = " << subtree_height << std::endl;
    std::cout << "Cutoff height = " << height - subtree_height << std::endl;
    std::cout << "Number of leaves in a subtree = " << subtree_leaves_size << std::endl;
    std::cout << "State of a subtree = " << subtree_states_size << std::endl;
    std::cout << "Digest elements for a subtree = " << get_digests_len<S>(subtree_height, tree_config.arity)
              << std::endl;
    std::cout << "Size of 1 subtree states = " << subtree_states_size * sizeof(S) / 1024 / 1024 << " MB" << std::endl;
    std::cout << "Size of 1 subtree digests = " << subtree_digests_size * sizeof(S) / 1024 / 1024 << " MB" << std::endl;
    std::cout << "Cap height" << cap_height << std::endl;
    std::cout << "Enabling caps mode? " << caps_mode << std::endl;
#endif

    // Build subtrees in parallel. This for loop invokes kernels that can run in a pool of size `number_of_streams`
    for (size_t subtree_idx = 0; subtree_idx < number_of_subtrees; subtree_idx++) {
      size_t stream_idx = subtree_idx % number_of_streams;
      hipStream_t subtree_stream = streams[stream_idx];

      const L* subtree_leaves = leaves + subtree_idx * subtree_leaves_size * sponge_config.input_block_len;
      D* subtree_state = states_ptr + stream_idx * subtree_states_size;
      D* subtree_digests = digests_ptr + stream_idx * subtree_digests_size;

      int subtree_keep_rows = 0;
      if (config.keep_rows) {
        int diff = config.keep_rows - cap_height;
        subtree_keep_rows = diff <= 0 ? 1 : diff;
      }
      hipError_t subtree_result = build_merkle_subtree<L, D>(
        subtree_leaves,             // leaves
        subtree_state,              // state
        subtree_digests,            // digests
        subtree_idx,                // subtree_idx
        subtree_height,             // subtree_height
        caps_mode ? caps : digests, // big_tree_digests
        number_of_leaves,           // start_segment_size
        0,                          // start_segment_offset
        subtree_keep_rows,          // keep_rows
        sponge,                     // hash
        compression,                // hash
        sponge_config,              // hash
        subtree_stream              // stream
      );
      CHK_IF_RETURN(subtree_result);
    }

    for (size_t i = 0; i < number_of_streams; i++) {
      CHK_IF_RETURN(hipStreamSynchronize(streams[i]));
    }

    // Finish the top-level tree if any
    // if (cap_height > 0) {
    //   size_t start_segment_size = caps_len / tree_config.arity;
    //   size_t start_segment_offset = 0;
    //   if (!caps_mode) {
    //     size_t layer_size = pow(tree_config.arity, config.keep_rows - 1);
    //     for (int i = 0; i < config.keep_rows - cap_height + 1; i++) {
    //       start_segment_offset += layer_size;
    //       layer_size /= tree_config.arity;
    //     }
    //   }
    //   CHK_IF_RETURN(hipMemcpy2DAsync(
    //     states_ptr, T * sizeof(S), caps_mode ? caps : (digests + start_segment_offset - caps_len), tree_config.arity
    //     * sizeof(S), tree_config.arity * sizeof(S), caps_len / tree_config.arity,                 // Size of the
    //     source hipMemcpyHostToDevice, stream)); // Direction and stream

    //   hipError_t top_tree_result = build_merkle_subtree<S, T>(
    //     states_ptr,           // state
    //     digests_ptr,          // digests
    //     0,                    // subtree_idx
    //     cap_height,           // subtree_height
    //     digests,              // big_tree_digests
    //     start_segment_size,   // start_segment_size
    //     start_segment_offset, // start_segment_offset
    //     config.keep_rows,     // keep_rows
    //     poseidon,             // hash
    //     stream                // stream
    //   );
    //   CHK_IF_RETURN(top_tree_result);
    //   if (caps_mode) { free(caps); }
    // }

    CHK_IF_RETURN(hipFreeAsync(states_ptr, stream));
    CHK_IF_RETURN(hipFreeAsync(digests_ptr, stream));
    if (!config.is_async) return CHK_STICKY(hipStreamSynchronize(stream));
    for (size_t i = 0; i < number_of_streams; i++) {
      CHK_IF_RETURN(hipStreamSynchronize(streams[i]));
      CHK_IF_RETURN(hipStreamDestroy(streams[i]));
    }
    free(streams);
    return CHK_LAST();
  }

} // namespace merkle_tree