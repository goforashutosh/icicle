#include "utils/utils.h"

#include "fields/field_config.cuh"
using namespace field_config;

#include "gpu-utils/error_handler.cuh"
#include "merkle-tree/merkle.cuh"

#include "poseidon2/poseidon2.cuh"

namespace merkle_tree {
  extern "C" hipError_t CONCAT_EXPAND(FIELD, build_poseidon2_merkle_tree)(
    const scalar_t* leaves_digests,
    scalar_t* digests,
    uint32_t height,
    int arity,
    Poseidon2<scalar_t>& poseidon,
    TreeBuilderConfig& config)
  {
    return build_merkle_tree<scalar_t>(leaves_digests, digests, height, poseidon, config);
  }
} // namespace merkle_tree