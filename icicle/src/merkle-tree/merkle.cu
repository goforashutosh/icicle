#include "hip/hip_runtime.h"
#include "hash/hash.cuh"
#include "merkle-tree/merkle.cuh"

namespace merkle_tree {
  /// Constructs merkle subtree without parallelization
  /// The digests are aligned sequentially per row
  /// Example:
  ///
  /// Big tree:
  ///
  ///        1      <- Root
  ///       / \     <- Arity = 2
  ///      2   3    <- Digests
  ///     / \ / \   <- Height = 2 (as the number of edges)
  ///    4  5 6  7  <- height^arity leaves
  ///    |  | |  |  <- Sponge hash 1 to 1
  ///    a  b c  d  <- Input vector 1x4
  ///
  /// Subtree 1    Subtree 2
  ///    2            3
  ///   / \          / \
  ///  4   5        6   7
  ///
  /// Digests array for subtree 1:
  /// [4 5 . . 2 . .]
  /// |   |    |
  /// -----    V
  ///   |    Segment (offset = 4, subtree_idx = 0)
  ///   v
  /// Segment (offset = 0, subtree_idx = 0)
  ///
  /// Digests array for subtree 2:
  /// [. . 6 7 . 3 .]
  ///     |   |
  ///     -----
  ///       |
  ///       v
  ///    Segment (offset = 0, subtree_idx = 1)
  ///
  /// Total digests array:
  /// [4 5 6 7 2 3 .]
  template <typename H, typename L, typename D>
  hipError_t build_merkle_subtree(
    const L* leaves,
    D* states,
    D* digests,
    size_t subtree_idx,
    size_t subtree_height,
    L* big_tree_digests,
    size_t start_segment_size,
    size_t start_segment_offset,
    unsigned int keep_rows,
    unsigned int input_block_len, 
    const SpongeHasher<H, L, D>& sponge,
    const CompressionHasher<H, D>& compression,
    const SpongeConfig& sponge_config,
    hipStream_t& stream)
  {
    unsigned int arity = compression.get_preimage_max_length();

    SpongeConfig local_sponge_config = sponge_config;
    local_sponge_config.recursive_squeeze = true;

    size_t leaves_size = pow(arity, subtree_height);

    std::cout << "Absorbing" << std::endl;
    sponge.absorb_many(leaves, states, leaves_size, input_block_len, local_sponge_config);
    std::cout << "Squeezing" << std::endl;
    sponge.squeeze_many(states, digests, leaves_size, 1, local_sponge_config);

    uint32_t number_of_states = leaves_size;
    size_t segment_size = start_segment_size;
    size_t segment_offset = start_segment_offset;

    device_context::DeviceContext subtree_ctx{
      stream,
      sponge_config.ctx.device_id,
      sponge_config.ctx.mempool,
    };

    while (number_of_states > 0) {
      std::cout << "Compressing " << number_of_states << std::endl;
      hipError_t compression_error =
        compression.compress_many(digests, states, number_of_states, sponge_config.offset, subtree_ctx);
      CHK_IF_RETURN(compression_error);

      std::cout << "Compressed " << number_of_states << std::endl;
      if (!keep_rows || subtree_height <= keep_rows) {
        D* digests_with_offset = big_tree_digests + segment_offset + subtree_idx * number_of_states;
        CHK_IF_RETURN(
          hipMemcpyAsync(digests_with_offset, states, number_of_states * sizeof(D), hipMemcpyDeviceToHost, stream));
        segment_offset += segment_size;
      }

      segment_size /= arity;
      subtree_height--;
      number_of_states /= arity;

      D* tmp = states;
      states = digests;
      digests = tmp;
    }

    return CHK_LAST();
  }

  template <typename H, typename L, typename D>
  hipError_t build_merkle_tree(
    const L* leaves,
    D* digests,
    unsigned int height,
    unsigned int arity,
    unsigned int input_block_len, 
    const CompressionHasher<H, D>& compression,
    const SpongeHasher<H, L, D>& sponge,
    const SpongeConfig& sponge_config,
    const TreeBuilderConfig& tree_config)
  {
    CHK_INIT_IF_RETURN();
    hipStream_t& stream = tree_config.ctx.stream;

    if (input_block_len > sponge_config.input_rate)
      THROW_ICICLE_ERR(
        IcicleError_t::InvalidArgument,
        "Sponge construction at the bottom of the tree doesn't support inputs bigger than hash rate");
    if (compression.get_preimage_max_length() != tree_config.arity)
      THROW_ICICLE_ERR(IcicleError_t::InvalidArgument, "Hash max preimage length does not match merkle tree arity");

    uint32_t number_of_leaves = pow(tree_config.arity, height);
    uint32_t total_number_of_leaves = number_of_leaves * input_block_len;

    // This will determine how much splitting do we need to do
    // `number_of_streams` subtrees should fit in the device
    // This means each subtree should fit in `STREAM_CHUNK_SIZE` memory
    uint32_t number_of_subtrees = 1;
    uint32_t subtree_height = height;
    uint32_t subtree_leaves_size = number_of_leaves;
    uint32_t subtree_states_size = subtree_leaves_size * sponge.get_width();
    uint32_t subtree_digests_size = subtree_states_size * sponge.get_preimage_max_length() / sponge.get_width();
    size_t subtree_memory_required = sizeof(D) * (subtree_states_size + subtree_digests_size);
    while (subtree_memory_required > STREAM_CHUNK_SIZE) {
      number_of_subtrees *= tree_config.arity;
      subtree_height--;
      subtree_leaves_size /= tree_config.arity;
      subtree_states_size /= tree_config.arity;
      subtree_digests_size /= tree_config.arity;
      subtree_memory_required = sizeof(D) * (subtree_states_size + subtree_digests_size);
    }
    int cap_height = height - subtree_height;
    size_t caps_len = pow(tree_config.arity, cap_height);

    size_t available_memory, _total_memory;
    CHK_IF_RETURN(hipMemGetInfo(&available_memory, &_total_memory));
    available_memory -= GIGA / 8; // Leave 128 MB just in case

    // We can effectively parallelize memory copy with streams
    // as long as they don't operate on more than `STREAM_CHUNK_SIZE` bytes
    const size_t number_of_streams = std::min((uint32_t)(available_memory / STREAM_CHUNK_SIZE), number_of_subtrees);
    hipStream_t* streams = static_cast<hipStream_t*>(malloc(sizeof(hipStream_t) * number_of_streams));
    for (size_t i = 0; i < number_of_streams; i++) {
      CHK_IF_RETURN(hipStreamCreate(&streams[i]));
    }

    // Allocate memory for the leaves and digests
    // These are shared by streams in a pool
    D *states_ptr, *digests_ptr;
    CHK_IF_RETURN(hipMallocAsync(&states_ptr, subtree_states_size * number_of_streams * sizeof(D), stream))
    CHK_IF_RETURN(hipMallocAsync(&digests_ptr, subtree_digests_size * number_of_streams * sizeof(D), stream))
    // Wait for these allocations to finish
    CHK_IF_RETURN(hipStreamSynchronize(stream));

    bool caps_mode = tree_config.keep_rows && tree_config.keep_rows < cap_height;
    D* caps;
    if (caps_mode) { caps = static_cast<D*>(malloc(caps_len * sizeof(D))); }

#ifdef MERKLE_DEBUG
    std::cout << "Available memory = " << available_memory / 1024 / 1024 << " MB" << std::endl;
    std::cout << "Number of streams = " << number_of_streams << std::endl;
    std::cout << "Number of subtrees = " << number_of_subtrees << std::endl;
    std::cout << "Height of a subtree = " << subtree_height << std::endl;
    std::cout << "Cutoff height = " << height - subtree_height << std::endl;
    std::cout << "Number of leaves in a subtree = " << subtree_leaves_size << std::endl;
    std::cout << "State of a subtree = " << subtree_states_size << std::endl;
    std::cout << "Digest elements for a subtree = " << subtree_digests_size << std::endl;
    std::cout << "Size of 1 subtree states = " << subtree_states_size * sizeof(D) / 1024 / 1024 << " MB" << std::endl;
    std::cout << "Size of 1 subtree digests = " << subtree_digests_size * sizeof(D) / 1024 / 1024 << " MB" << std::endl;
    std::cout << "Cap height = " << cap_height << std::endl;
    std::cout << "Enabling caps mode? " << caps_mode << std::endl;
#endif

    // Build subtrees in parallel. This for loop invokes kernels that can run in a pool of size `number_of_streams`
    for (size_t subtree_idx = 0; subtree_idx < number_of_subtrees; subtree_idx++) {
      std::cout << "Processing tree #" << subtree_idx << std::endl;
      size_t stream_idx = subtree_idx % number_of_streams;
      hipStream_t subtree_stream = streams[stream_idx];

      const L* subtree_leaves = leaves + subtree_idx * subtree_leaves_size * input_block_len;
      D* subtree_state = states_ptr + stream_idx * subtree_states_size;
      D* subtree_digests = digests_ptr + stream_idx * subtree_digests_size;

      int subtree_keep_rows = 0;
      if (tree_config.keep_rows) {
        int diff = tree_config.keep_rows - cap_height;
        subtree_keep_rows = diff <= 0 ? 1 : diff;
      }
      hipError_t subtree_result = build_merkle_subtree<H, L, D>(
        subtree_leaves,             // leaves
        subtree_state,              // state
        subtree_digests,            // digests
        subtree_idx,                // subtree_idx
        subtree_height,             // subtree_height
        caps_mode ? caps : digests, // big_tree_digests
        number_of_leaves,           // start_segment_size
        0,                          // start_segment_offset
        subtree_keep_rows,          // keep_rows
        input_block_len,            // input_block_len
        sponge,                     // hash
        compression,                // hash
        sponge_config,              // hash
        subtree_stream              // stream
      );
      CHK_IF_RETURN(subtree_result);
    }

    for (size_t i = 0; i < number_of_streams; i++) {
      CHK_IF_RETURN(hipStreamSynchronize(streams[i]));
    }

    // Finish the top-level tree if any
    // if (cap_height > 0) {
    //   size_t start_segment_size = caps_len / tree_config.arity;
    //   size_t start_segment_offset = 0;
    //   if (!caps_mode) {
    //     size_t layer_size = pow(tree_config.arity, config.keep_rows - 1);
    //     for (int i = 0; i < config.keep_rows - cap_height + 1; i++) {
    //       start_segment_offset += layer_size;
    //       layer_size /= tree_config.arity;
    //     }
    //   }
    //   CHK_IF_RETURN(hipMemcpy2DAsync(
    //     states_ptr, T * sizeof(S), caps_mode ? caps : (digests + start_segment_offset - caps_len), tree_config.arity
    //     * sizeof(S), tree_config.arity * sizeof(S), caps_len / tree_config.arity,                 // Size of the
    //     source hipMemcpyHostToDevice, stream)); // Direction and stream

    //   hipError_t top_tree_result = build_merkle_subtree<S, T>(
    //     states_ptr,           // state
    //     digests_ptr,          // digests
    //     0,                    // subtree_idx
    //     cap_height,           // subtree_height
    //     digests,              // big_tree_digests
    //     start_segment_size,   // start_segment_size
    //     start_segment_offset, // start_segment_offset
    //     config.keep_rows,     // keep_rows
    //     poseidon,             // hash
    //     stream                // stream
    //   );
    //   CHK_IF_RETURN(top_tree_result);
    //   if (caps_mode) { free(caps); }
    // }

    CHK_IF_RETURN(hipFreeAsync(states_ptr, stream));
    CHK_IF_RETURN(hipFreeAsync(digests_ptr, stream));
    if (!tree_config.is_async) return CHK_STICKY(hipStreamSynchronize(stream));
    for (size_t i = 0; i < number_of_streams; i++) {
      CHK_IF_RETURN(hipStreamSynchronize(streams[i]));
      CHK_IF_RETURN(hipStreamDestroy(streams[i]));
    }
    free(streams);
    return CHK_LAST();
  }

} // namespace merkle_tree