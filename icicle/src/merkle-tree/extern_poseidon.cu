#include "utils/utils.h"

#include "gpu-utils/error_handler.cuh"
#include "merkle-tree/merkle.cuh"
#include "merkle.cu"

#include "hash/hash.cuh"
#include "poseidon/poseidon.cuh"

#include "fields/field_config.cuh"
using namespace field_config;

namespace merkle_tree {
  extern "C" hipError_t CONCAT_EXPAND(FIELD, build_poseidon_merkle_tree)(
    const scalar_t* leaves_digests,
    scalar_t* digests,
    unsigned int height,
    unsigned int input_block_len,
    const poseidon::Poseidon<scalar_t>* poseidon_compression,
    const poseidon::Poseidon<scalar_t>* poseidon_sponge,
    const TreeBuilderConfig& tree_config)
  {
    return build_merkle_tree<scalar_t, scalar_t>(
      leaves_digests, digests, height, input_block_len, *poseidon_compression, *poseidon_sponge, tree_config);
  }
} // namespace merkle_tree