#include <cstdint>
#include "gpu-utils/device_context.cuh"
#include "gpu-utils/error_handler.cuh"

#include "hash/hash.cuh"
#include "hash/keccak/keccak.cuh"
#include "kernels.cu"

using namespace hash;

namespace keccak {
  hipError_t Keccak::pad_many(
    uint64_t* states,
    unsigned int number_of_states,
    unsigned int input_block_len,
    const device_context::DeviceContext& ctx) const
  {
    unsigned int input_len = input_block_len % this->rate;
    keccak_10_1_pad_kernel<<<keccak_number_of_blocks(number_of_states), KECCAK_BLOCK_SIZE, 0, ctx.stream>>>(
      states, input_len, this->rate, number_of_states);

    CHK_IF_RETURN(hipPeekAtLastError());
    return CHK_LAST();
  }

  hipError_t Keccak::squeeze_states(
    const uint64_t* states,
    unsigned int number_of_states,
    unsigned int output_len,
    uint64_t* output,
    const device_context::DeviceContext& ctx) const
  {
    switch (this->rate) {
    case 17:
      squeeze_states_kernel<4><<<keccak_number_of_blocks(number_of_states), KECCAK_BLOCK_SIZE, 0, ctx.stream>>>(
        states, number_of_states, output);
      break;
    case 9:
      squeeze_states_kernel<8><<<keccak_number_of_blocks(number_of_states), KECCAK_BLOCK_SIZE, 0, ctx.stream>>>(
        states, number_of_states, output);
      break;
    default:
      THROW_ICICLE_ERR(IcicleError_t::InvalidArgument, "KeccakSqueeze: #rate must be one of [17, 9]");
    }

    CHK_IF_RETURN(hipPeekAtLastError());
    return CHK_LAST();
  }

  hipError_t Keccak::run_permutation_kernel(
    const uint64_t* states,
    uint64_t* output,
    unsigned int number_of_states,
    bool aligned,
    const device_context::DeviceContext& ctx) const
  {
    keccak_permutation_kernel<<<keccak_number_of_blocks(number_of_states), KECCAK_BLOCK_SIZE, 0, ctx.stream>>>(
      states, output, number_of_states);

    CHK_IF_RETURN(hipPeekAtLastError());
    return CHK_LAST();
  }

  template <int C, int D>
  hipError_t
  keccak_hash(uint8_t* input, int input_block_size, int number_of_blocks, uint8_t* output, KeccakConfig& config)
  {
    CHK_INIT_IF_RETURN();
    hipStream_t& stream = config.ctx.stream;

    uint8_t* input_device;
    if (config.are_inputs_on_device) {
      input_device = input;
    } else {
      CHK_IF_RETURN(hipMallocAsync(&input_device, number_of_blocks * input_block_size, stream));
      CHK_IF_RETURN(
        hipMemcpyAsync(input_device, input, number_of_blocks * input_block_size, hipMemcpyHostToDevice, stream));
    }

    uint8_t* output_device;
    if (config.are_outputs_on_device) {
      output_device = output;
    } else {
      CHK_IF_RETURN(hipMallocAsync(&output_device, number_of_blocks * (D / 8), stream));
    }

    int number_of_threads = 512;
    int number_of_gpu_blocks = (number_of_blocks - 1) / number_of_threads + 1;
    keccak_hash_blocks<C, D><<<number_of_gpu_blocks, number_of_threads, 0, stream>>>(
      input_device, input_block_size, number_of_blocks, output_device);

    if (!config.are_inputs_on_device) CHK_IF_RETURN(hipFreeAsync(input_device, stream));

    if (!config.are_outputs_on_device) {
      CHK_IF_RETURN(hipMemcpyAsync(output, output_device, number_of_blocks * (D / 8), hipMemcpyDeviceToHost, stream));
      CHK_IF_RETURN(hipFreeAsync(output_device, stream));
    }

    if (!config.is_async) return CHK_STICKY(hipStreamSynchronize(stream));
    return CHK_LAST();
  }
} // namespace keccak