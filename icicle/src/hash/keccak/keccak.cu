#include <cstdint>
#include "gpu-utils/device_context.cuh"
#include "gpu-utils/error_handler.cuh"

#include "hash/hash.cuh"
#include "hash/keccak/keccak.cuh"
#include "kernels.cu"

using namespace hash;

namespace keccak {
  hipError_t Keccak::pad_many(
    uint64_t* states,
    unsigned int number_of_states,
    unsigned int input_block_len,
    const device_context::DeviceContext& ctx) const
  {
    unsigned int input_len = input_block_len % this->rate;
    keccak_10_1_pad_kernel<<<keccak_number_of_blocks(number_of_states), KECCAK_BLOCK_SIZE, 0, ctx.stream>>>(
      states, input_len, this->rate, number_of_states);

    CHK_IF_RETURN(hipPeekAtLastError());
    return CHK_LAST();
  }

  hipError_t Keccak::squeeze_states(
    const uint64_t* states,
    unsigned int number_of_states,
    unsigned int output_len,
    uint64_t* output,
    const device_context::DeviceContext& ctx) const
  {
    switch (this->rate) {
    case 17:
      squeeze_states_kernel<4><<<keccak_number_of_blocks(number_of_states), KECCAK_BLOCK_SIZE, 0, ctx.stream>>>(
        states, number_of_states, output);
      break;
    case 9:
      squeeze_states_kernel<8><<<keccak_number_of_blocks(number_of_states), KECCAK_BLOCK_SIZE, 0, ctx.stream>>>(
        states, number_of_states, output);
      break;
    default:
      THROW_ICICLE_ERR(IcicleError_t::InvalidArgument, "KeccakSqueeze: #rate must be one of [17, 9]");
    }

    CHK_IF_RETURN(hipPeekAtLastError());
    return CHK_LAST();
  }

  hipError_t Keccak::run_permutation_kernel(
    const uint64_t* states,
    uint64_t* output,
    unsigned int number_of_states,
    bool aligned,
    const device_context::DeviceContext& ctx) const
  {
    keccak_permutation_kernel<<<keccak_number_of_blocks(number_of_states), KECCAK_BLOCK_SIZE, 0, ctx.stream>>>(
      states, output, number_of_states);

    CHK_IF_RETURN(hipPeekAtLastError());
    return CHK_LAST();
  }
} // namespace keccak