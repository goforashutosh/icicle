#include "utils/utils.h"

#include "fields/field_config.cuh"
using namespace field_config;

#include "gpu-utils/error_handler.cuh"
#include "poseidon2/poseidon2.cuh"
#include "poseidon2.cu"

namespace poseidon2 {
  template class Poseidon2<scalar_t>;

  extern "C" hipError_t CONCAT_EXPAND(FIELD, poseidon2_create_cuda)(
    Poseidon2<scalar_t>** poseidon,
    unsigned int width,
    unsigned int alpha,
    unsigned int internal_rounds,
    unsigned int external_rounds,
    const scalar_t* round_constants,
    const scalar_t* internal_matrix_diag,
    MdsType mds_type,
    DiffusionStrategy diffusion,
    device_context::DeviceContext& ctx)
  {
    try {
      *poseidon = new Poseidon2<scalar_t>(
        width, alpha, internal_rounds, external_rounds, round_constants, internal_matrix_diag, mds_type, diffusion,
        ctx);
      return hipError_t::hipSuccess;
    } catch (const IcicleError& _error) {
      return hipError_t::hipErrorUnknown;
    }
  }

  extern "C" hipError_t CONCAT_EXPAND(FIELD, poseidon2_load_cuda)(
    Poseidon2<scalar_t>** poseidon,
    unsigned int width,
    MdsType mds_type,
    DiffusionStrategy diffusion,
    device_context::DeviceContext& ctx)
  {
    try {
      *poseidon = new Poseidon2<scalar_t>(width, mds_type, diffusion, ctx);
      return hipError_t::hipSuccess;
    } catch (const IcicleError& _error) {
      return hipError_t::hipErrorUnknown;
    }
  }

  extern "C" hipError_t CONCAT_EXPAND(FIELD, poseidon2_permute_many_cuda)(
    const Poseidon2<scalar_t>* poseidon,
    const scalar_t* states,
    scalar_t* output,
    unsigned int number_of_states,
    device_context::DeviceContext& ctx,
    bool is_async)
  {
    return poseidon->permute_many(states, output, number_of_states, ctx, is_async);
  }

  extern "C" hipError_t CONCAT_EXPAND(FIELD, poseidon2_compress_many_cuda)(
    const Poseidon2<scalar_t>* poseidon,
    const scalar_t* states,
    scalar_t* output,
    unsigned int number_of_states,
    unsigned int offset,
    scalar_t* perm_output,
    device_context::DeviceContext& ctx,
    bool is_async)
  {
    return poseidon->compress_many(states, output, number_of_states, offset, perm_output, ctx, is_async);
  }

  extern "C" hipError_t
  CONCAT_EXPAND(FIELD, poseidon2_delete_cuda)(Poseidon2<scalar_t>* poseidon, device_context::DeviceContext& ctx)
  {
    try {
      poseidon->~Poseidon2();
      return hipError_t::hipSuccess;
    } catch (const IcicleError& _error) {
      return hipError_t::hipErrorUnknown;
    }
  }
} // namespace poseidon2